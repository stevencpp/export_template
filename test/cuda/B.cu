#include "hip/hip_runtime.h"
#include "B.h"
#include <B.xti>

#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <typeinfo>

namespace B {

	template < typename T >
	__global__ void addKernel(T *c, const T *a, const T *b)
	{
		int i = threadIdx.x;
		c[i] = a[i] + b[i];
	}

	template < typename T >
	bool addWithCuda(T *c, const T *a, const T *b, int size)
	{
		printf("%s -- T = %s\n", __FUNCSIG__, typeid(T).name());

		T *dev_a = 0;
		T *dev_b = 0;
		T *dev_c = 0;
		hipError_t cudaStatus;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(T));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(T));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(T));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_a, a, size * sizeof(T), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(dev_b, b, size * sizeof(T), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// Launch a kernel on the GPU with one thread for each element.
		addKernel << <1, size >> > (dev_c, dev_a, dev_b);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, dev_c, size * sizeof(T), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

	Error:
		hipFree(dev_c);

		hipFree(dev_a);
		hipFree(dev_b);

		return cudaStatus == hipSuccess;
	}

} // namespace B